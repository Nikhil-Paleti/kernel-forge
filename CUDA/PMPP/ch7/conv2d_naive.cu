#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdlib> 
#include <cmath> // For fabs
#include <vector> // For storing CPU output if you prefer
#include <iomanip> // For std::fixed and std::setprecision

void generateRandomMatrix(float *matrix, int rows, int cols, int seed=0){
    srand(seed);
    for (int i =0; i < rows*cols; i++){
        matrix[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

__global__ void _conv2d(float* N, float *F, float *P ,int height, int width, int r){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float acc = 0.0;
    int filter_dim = 2*r + 1;
    for (int i=0; i< filter_dim; i++){
        for (int j=0; j<filter_dim; j++){
            int local_x = x - r + i;
            int local_y = y - r + j;
            if ((0 <= local_x && local_x < width) && (0 <= local_y && local_y < height)){
                acc += N[local_y*width + local_x] * F[j*filter_dim + i];
            }
        }
    }
    if (x < width && y < height){
        P[y*width + x] = acc; 
    }
}

void conv2d(float *image, float *filter, float *output, int M, int N, int r){
    size_t image_size = M*N * sizeof(float);
    size_t filter_size = (2*r+1) * (2*r+1) * sizeof(float);
    float *image_d, *filter_d, *output_d;
    hipMalloc((void**)&image_d, image_size);
    hipMalloc((void**)&filter_d, filter_size);
    hipMalloc((void**)&output_d, image_size);
    hipMemcpy(image_d, image, image_size, hipMemcpyHostToDevice);
    hipMemcpy(filter_d, filter, filter_size, hipMemcpyHostToDevice);
    dim3 block(16, 16);
    dim3 grid((N + block.x - 1) / block.x , (M+block.y-1)/block.y);
    _conv2d<<<grid, block>>>(image_d, filter_d, output_d, M, N, r);
    hipMemcpy(output, output_d, image_size, hipMemcpyDeviceToHost);
    hipFree(image_d);
    hipFree(filter_d);
    hipFree(output_d);
}

void conv2d_cpu(const float *image, const float *filter, float *output, int height, int width, int r) {
    int filter_dim = 2 * r + 1;

    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x) {
            float acc = 0.0f;
            for (int j = 0; j < filter_dim; ++j) { // Filter row
                for (int i = 0; i < filter_dim; ++i) { // Filter col
                    int image_x = x - r + i;
                    int image_y = y - r + j;

                    if ((image_x >= 0 && image_x < width) && (image_y >= 0 && image_y < height)) {
                        acc += image[image_y * width + image_x] * filter[j * filter_dim + i];
                    }
                }
            }
            output[y * width + x] = acc;
        }
    }
}

int main(){
    int M = 1024;
    int N = 1024;
    int r = 1;
    int filter_dim_actual = (2*r + 1); // Renamed from filter_size to avoid conflict
    float *image = new float[M*N];
    float *filter = new float[ filter_dim_actual * filter_dim_actual];
    float *output_gpu = new float[M*N]; // Output from GPU
    float *output_cpu = new float[M*N]; // Output from CPU

    generateRandomMatrix(image, M, N, 0); // Use fixed seeds for reproducibility
    generateRandomMatrix(filter, filter_dim_actual, filter_dim_actual, 1);

    // GPU Convolution
    conv2d(image, filter, output_gpu, M, N, r);

    // CPU Convolution
    conv2d_cpu(image, filter, output_cpu, M, N, r);

    // Comparison
    float epsilon = 1e-5f; // Tolerance for floating point comparison
    bool mismatch_found = false;
    for (int i = 0; i < M * N; ++i) {
        if (std::fabs(output_gpu[i] - output_cpu[i]) > epsilon) {
            std::cerr << "Mismatch found at index " << i << "!" << std::endl;
            std::cerr << "GPU output: " << std::fixed << std::setprecision(8) << output_gpu[i] << std::endl;
            std::cerr << "CPU output: " << std::fixed << std::setprecision(8) << output_cpu[i] << std::endl;
            std::cerr << "Difference: " << std::fabs(output_gpu[i] - output_cpu[i]) << std::endl;
            mismatch_found = true;
            // You might want to break here or limit the number of printed mismatches
            // break; 
        }
    }

    if (!mismatch_found) {
        std::cout << "Verification successful! GPU and CPU outputs match within tolerance." << std::endl;
    } else {
        std::cout << "Verification failed! GPU and CPU outputs differ." << std::endl;
    }

    delete[] image;
    delete[] filter;
    delete[] output_gpu;
    delete[] output_cpu;
    return 0;
}