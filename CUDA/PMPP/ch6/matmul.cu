#include <iostream> 
#include <hip/hip_runtime.h>
#include <cstdlib> 

#define TILE_WIDTH 16
#define COARSE_FACTOR 2

void generateRandomMatrix(float *matrix, int rows, int cols){
    for(int i=0; i<rows*cols; i++){
        matrix[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

__global__ void _matmul(float *A, float *B, float *C, int M, int N, int K){
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int x = (blockIdx.x * COARSE_FACTOR) * blockDim.x  + tx;
    int y = blockIdx.y * blockDim.y + ty; 

    float acc[COARSE_FACTOR];
    for (int c=0; c < COARSE_FACTOR; c++){
        acc[c] = 0.0;
    }

    for (int ph=0; ph < (K + TILE_WIDTH - 1) / (TILE_WIDTH); ph++){
        if (ph*TILE_WIDTH + tx < K && y < M ){
            Mds[ty][tx] = A[y*K + ph*TILE_WIDTH + tx];
        }else{
            Mds[ty][tx] = 0.0;
        }

        for(int c=0; c< COARSE_FACTOR; c++){
            if (ph*TILE_WIDTH + ty < K && x + TILE_WIDTH*c < N){
                Nds[ty][tx] = B[(ph*TILE_WIDTH + ty) * N + (x + TILE_WIDTH * c)];
            }else{
                Nds[ty][tx] = 0.0;
            }
            __syncthreads();

            for (int k =0; k < TILE_WIDTH; k++){
                acc[c] += Mds[ty][k] * Nds[k][tx];
            }
            __syncthreads();

        }

    }
    for (int c=0; c < COARSE_FACTOR; c++){
        if (x + c*TILE_WIDTH < N && y < M){
            C[ y*N + x + c*TILE_WIDTH] = acc[c]; 
        }
    }
}

void matmul(float *A_h, float *B_h, float *C_h, int M, int N, int K){
    size_t size_A = M*K * sizeof(float);
    size_t size_B = K*N * sizeof(float);
    size_t size_C = M*N * sizeof(float);
    float *A_d, *B_d, *C_d;
    hipMalloc((void**)&A_d, size_A);
    hipMalloc((void**)&B_d, size_B);
    hipMalloc((void**)&C_d, size_C);
    hipMemcpy(A_d, A_h, size_A, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size_B, hipMemcpyHostToDevice);
    dim3 block(TILE_WIDTH, TILE_WIDTH);
    dim3 grid((N + TILE_WIDTH*COARSE_FACTOR -1)/ (TILE_WIDTH*COARSE_FACTOR), (M + TILE_WIDTH -1)/ TILE_WIDTH);
    _matmul<<<grid, block>>>(A_d, B_d, C_d, M, N, K);
    hipMemcpy(C_h, C_d, size_C, hipMemcpyDeviceToHost);
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main(){
    int M = 1024;
    int N = 1024;
    int K = 768; 
    float *A_h = new float[M*K];
    float *B_h = new float[K*N];
    float *C_h = new float[M*N];
    generateRandomMatrix(A_h, M, K);
    generateRandomMatrix(B_h, K, N);
    matmul(A_h, B_h, C_h, M, N, K);
    delete[] A_h;
    delete[] B_h;
    delete[] C_h;
    return 0;
}